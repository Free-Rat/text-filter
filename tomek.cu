#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>

#define DEFAULT_MAIN_FILE "main.txt"
#define DEFAULT_REMOVE_FILE "remove.txt"
#define DEFAULT_OUTPUT_FILE "output.txt"

__device__ bool device_strncmp(const char* str1, const char* str2, int len) {
    for (int i = 0; i < len; i++) {
        if (str1[i] != str2[i]) {
            return false;
        }
    }
    return true;
}

__global__ void removeWordsKernel(const char* text, const char** words, const int* wordLens, int numWords, int textLen, char* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < textLen) {
        bool isWord = false;
        for (int i = 0; i < numWords; i++) {
            int wordLen = wordLens[i];
            if (idx + wordLen <= textLen && device_strncmp(&text[idx], words[i], wordLen)) {
                for (int j = 0; j < wordLen; j++) {
                    result[idx + j] = ' ';
                }
                idx += wordLen - 1; // Move index to the end of the word
                isWord = true;
                break;
            }
        }
        if (!isWord) {
            result[idx] = text[idx];
        }
    }
}

void readFileToString(const std::string& filename, std::string& data) {
    std::ifstream file(filename);
    if (file.is_open()) {
        std::stringstream buffer;
        buffer << file.rdbuf();
        data = buffer.str();
    } else {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
}

std::vector<std::string> readFileToVector(const std::string& filename) {
    std::ifstream file(filename);
    std::vector<std::string> lines;
    std::string line;
    if (file.is_open()) {
        while (getline(file, line)) {
            lines.push_back(line);
        }
    } else {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
    return lines;
}

int main(int argc, char* argv[]) {
    std::string mainFile = (argc > 1) ? argv[1] : DEFAULT_MAIN_FILE;
    std::string removeFile = (argc > 2) ? argv[2] : DEFAULT_REMOVE_FILE;
    std::string outputFile = (argc > 3) ? argv[3] : DEFAULT_OUTPUT_FILE;

    std::string text;
    readFileToString(mainFile, text);
    std::vector<std::string> removeWords = readFileToVector(removeFile);

    int textLen = text.size();
    int numWords = removeWords.size();
    std::vector<int> wordLens(numWords);

    for (int i = 0; i < numWords; i++) {
        wordLens[i] = removeWords[i].length();
    }

    // Prepare text and words for GPU
    char* d_text;
    char** d_words;
    int* d_wordLens;
    char* d_result;
    hipMalloc((void**)&d_text, textLen * sizeof(char));
    hipMalloc((void**)&d_words, numWords * sizeof(char*));
    hipMalloc((void**)&d_wordLens, numWords * sizeof(int));
    hipMalloc((void**)&d_result, textLen * sizeof(char));

    hipMemcpy(d_text, text.c_str(), textLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLens, wordLens.data(), numWords * sizeof(int), hipMemcpyHostToDevice);

    // Allocate and copy words to device
    char** h_words = (char**)malloc(numWords * sizeof(char*));
    for (int i = 0; i < numWords; i++) {
        hipMalloc((void**)&h_words[i], wordLens[i] * sizeof(char));
        hipMemcpy(h_words[i], removeWords[i].c_str(), wordLens[i] * sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_words, h_words, numWords * sizeof(char*), hipMemcpyHostToDevice);

    // Initialize result on the device with the original text
    hipMemcpy(d_result, d_text, textLen * sizeof(char), hipMemcpyDeviceToDevice);

    int blockSize = 256;
    int numBlocks = (textLen + blockSize - 1) / blockSize;

    removeWordsKernel<<<numBlocks, blockSize>>>(d_text, (const char**)d_words, d_wordLens, numWords, textLen, d_result);

    hipDeviceSynchronize(); // Ensure the kernel has completed

    char* result = new char[textLen + 1];
    result[textLen] = '\0'; // Null-terminate the result string
    hipMemcpy(result, d_result, textLen * sizeof(char), hipMemcpyDeviceToHost);

    std::ofstream outFile(outputFile);
    if (outFile.is_open()) {
        outFile.write(result, textLen);
        outFile.close();
    } else {
        std::cerr << "Error opening output file: " << outputFile << std::endl;
    }

    // Free allocated memory
    hipFree(d_text);
    hipFree(d_words);
    hipFree(d_wordLens);
    hipFree(d_result);
    for (int i = 0; i < numWords; i++) {
        hipFree(h_words[i]);
    }
    free(h_words);
    delete[] result;

    return 0;
}
